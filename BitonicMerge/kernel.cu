#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>

#define MAX_THREADS_PER_BLOCK 1024

// Bitonic Sort for CPU
void bitonicSortCPU(int* arr, int n) 
{
    for (int k = 2; k <= n; k *= 2) 
    {
        for (int j = k / 2; j > 0; j /= 2) 
        {
            for (int i = 0; i < n; i++) 
            {
                int ij = i ^ j;

                if (ij > i) 
                {
                    if ((i & k) == 0) 
                    {
                        if (arr[i] > arr[ij])
                        {
                            int temp = arr[i];
                            arr[i] = arr[ij];
                            arr[ij] = temp;
                        }
                    }
                    else 
                    {
                        if (arr[i] < arr[ij])
                        {
                            int temp = arr[i];
                            arr[i] = arr[ij];
                            arr[ij] = temp;
                        }
                    }
                }
            }
        }
    }
}

//GPU Kernel Implementation of Bitonic Sort
__global__ void bitonicSortGPU(int* arr, int j, int k)
{
    unsigned int i, ij;

    i = threadIdx.x + blockDim.x * blockIdx.x;

    ij = i ^ j;

    if (ij > i)
    {
        if ((i & k) == 0)
        {
            if (arr[i] > arr[ij])
            {
                int temp = arr[i];
                arr[i] = arr[ij];
                arr[ij] = temp;
            }
        }
        else
        {
            if (arr[i] < arr[ij])
            {
                int temp = arr[i];
                arr[i] = arr[ij];
                arr[ij] = temp;
            }
        }
    }
}

//Device function for recursive Merge
__device__ void Merge(int* arr, int* temp, int left, int middle, int right) 
{
    int i = left;
    int j = middle;
    int k = left;

    while (i < middle && j < right) 
    {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }

    while (i < middle)
        temp[k++] = arr[i++];
    while (j < right)
        temp[k++] = arr[j++];

    for (int x = left; x < right; x++)
        arr[x] = temp[x];
}

//GPU Kernel for Merge Sort
__global__ void MergeSortGPU(int* arr, int* temp, int n, int width) 
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int left = tid * width;
    int middle = left + width / 2;
    int right = left + width;

    if (left < n && middle < n) 
    {
        Merge(arr, temp, left, middle, right);
    }
}

//CPU Merge Recursive Call function
void merge(int* arr, int* temp, int left, int mid, int right) 
{
    int i = left;
    int j = mid + 1;
    int k = left;

    while (i <= mid && j <= right) 
    {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }

    while (i <= mid)
        temp[k++] = arr[i++];

    while (j <= right)
        temp[k++] = arr[j++];

    for (int idx = left; idx <= right; ++idx)
        arr[idx] = temp[idx];
}

//CPU Implementation of Merge Sort
void mergeSortCPU(int* arr, int* temp, int left, int right) 
{
    if (left >= right)
        return;

    int mid = left + (right - left) / 2;

    mergeSortCPU(arr, temp, left, mid);
    mergeSortCPU(arr, temp, mid + 1, right);

    merge(arr, temp, left, mid, right);
}

//Function to print array
void printArray(int* arr, int size) 
{
    for (int i = 0; i < size; ++i)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

//Automated function to check if array is sorted
bool isSorted(int* arr, int size) 
{
    for (int i = 1; i < size; ++i) 
    {
        if (arr[i] < arr[i - 1])
            return false;
    }
    return true;
}

//Function to check if given number is a power of 2
bool isPowerOfTwo(int num) 
{
    return num > 0 && (num & (num - 1)) == 0;
}


//MAIN PROGRAM
int main()
{
    int choice;
    std::cout << "Select the type of sort:";
    std::cout << "\n\t1. Merge Sort";
    std::cout << "\n\t2. Bitonic Sort";
    std::cout << "\nEnter your choice: ";
    std::cin >> choice;

    
    if (choice < 1 || choice > 2)
    {
        while (choice != 1 || choice != 2)
        {
            std::cout << "\n!!!!! WRONG CHOICE. TRY AGAIN. YOU HAVE ONLY 2 DISTINCT OPTIONS-\n";
            std::cin >> choice;


            if (choice == 1 || choice == 2)
                break;
        }
    }

    if (choice == 1)
    {
        std::cout << "\n--------------------------------------------------------------\nMERGE SORT SELECTED\n--------------------------------------------------------------";
    }
    else
    {
        std::cout << "\n--------------------------------------------------------------\nBITONIC SORT SELECTED\n--------------------------------------------------------------";
    }

    int size;
    std::cout << "\n\nEnter the size of the array. Must be a power of 2:\n ";
    std::cin>>size;

    while (!isPowerOfTwo(size))
    {
        if (!isPowerOfTwo(size))
        {
            std::cout << "\nWrong Size, must be power of 2. Try again:\n ";
            std::cin>>size;
        }
        else
            break;
    }

    std::cout << "\n--------------------------------------------------------------\nSELECTED SORT PROCESS UNDERWAY\n--------------------------------------------------------------";
    
    //Create CPU based Arrays
    int* arr = new int[size];
    int* carr = new int[size];
    int* temp = new int[size];

    //Create GPU based arrays
    int* gpuArrmerge;
    int* gpuArrbiton;
    int* gpuTemp;

    // Initialize the array with random values
    srand(static_cast<unsigned int>(time(nullptr)));
    for (int i = 0; i < size; ++i) 
    {
        arr[i] = rand() % 100;
        carr[i] = arr[i];
    }

    //Print unsorted array 
    std::cout << "\n\nUnsorted array: ";
    if (size <= 100) 
    {
        printArray(arr, size);
    }
    else 
    {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }

    // Allocate memory on GPU
    hipMalloc((void**)&gpuArrmerge, size * sizeof(int));
    hipMalloc((void**)&gpuTemp, size * sizeof(int));
    hipMalloc((void**)&gpuArrbiton, size * sizeof(int));

    // Copy the input array to GPU memory
    hipMemcpy(gpuArrmerge, arr, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuArrbiton, arr, size * sizeof(int), hipMemcpyHostToDevice);

    // Perform GPU merge sort and measure time
    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    float millisecondsGPU = 0;

    //Initialize CPU clock counters
    clock_t startCPU, endCPU;

    //Set number of threads and blocks for kernel calls
    int threadsPerBlock = MAX_THREADS_PER_BLOCK;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    //Main If else block
    if (choice == 1)
    {
        //Call GPU Merge Kernel and time the run
        hipEventRecord(startGPU);
        for (int wid = 1; wid < size; wid *= 2)
        {
            MergeSortGPU << <threadsPerBlock, blocksPerGrid >> > (gpuArrmerge, gpuTemp, size, wid * 2);
        }
        hipEventRecord(stopGPU);

        //Transfer sorted array back to CPU
        hipMemcpy(arr, gpuArrmerge, size * sizeof(int), hipMemcpyDeviceToHost);

        //Calculate Elapsed GPU time
        hipEventSynchronize(stopGPU);
        hipEventElapsedTime(&millisecondsGPU, startGPU, stopGPU);

        //Time the CPU and call CPU Merge Sort
        startCPU = clock();
        mergeSortCPU(carr, temp, 0, size - 1);
        endCPU = clock();
    }

    else
    {
        int j, k;

        //Time the run and call GPU Bitonic Kernel
        hipEventRecord(startGPU);
        for (k = 2; k <= size; k <<= 1)
        {
            for (j = k >> 1; j > 0; j = j >> 1)
            {
                bitonicSortGPU << <blocksPerGrid, threadsPerBlock >> > (gpuArrbiton, j, k);
            }
        }
        hipEventRecord(stopGPU);

        //Transfer Sorted array back to CPU
        hipMemcpy(arr, gpuArrbiton, size * sizeof(int), hipMemcpyDeviceToHost);
        hipEventSynchronize(stopGPU);
        hipEventElapsedTime(&millisecondsGPU, startGPU, stopGPU);

        //Time the run and call CPU Bitonic Sort
        startCPU = clock();
        bitonicSortCPU(carr, size);
        endCPU = clock();
    }

    //Calculate Elapsed CPU time
    double millisecondsCPU = static_cast<double>(endCPU - startCPU) / (CLOCKS_PER_SEC / 1000.0);

    // Display sorted GPU array
    std::cout << "\n\nSorted GPU array: ";
    if (size <= 100) 
    {
        printArray(arr, size);
    }
    else {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }

    //Display sorted CPU array
    std::cout << "\nSorted CPU array: ";
    if (size <= 100) 
    {
        printArray(carr, size);
    }
    else {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }
    
    //Run the array with the automated isSorted checker
    if (isSorted(arr, size))
        std::cout << "\n\nSORT CHECKER RUNNING - SUCCESFULLY SORTED GPU ARRAY" << std::endl;
    else
        std::cout << "SORT CHECKER RUNNING - !!! FAIL !!!" << std::endl;
   
    if (isSorted(carr, size))
        std::cout << "SORT CHECKER RUNNING - SUCCESFULLY SORTED CPU ARRAY" << std::endl;
    else
        std::cout << "SORT CHECKER RUNNING - !!! FAIL !!!" << std::endl;

    //Print the time of the runs
    std::cout << "\n\nGPU Time: " << millisecondsGPU << " ms" << std::endl;
    std::cout << "CPU Time: " << millisecondsCPU << " ms" << std::endl;

    //Destroy all variables
    delete[] arr;
    delete[] carr;
    delete[] temp;

    //End
    hipFree(gpuArrmerge);
    hipFree(gpuArrbiton);
    hipFree(gpuTemp);

    std::cout << "\n------------------------------------------------------------------------------------\n||||| END. YOU MAY RUN THIS AGAIN |||||\n------------------------------------------------------------------------------------";
    return 0;
}